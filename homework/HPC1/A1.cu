

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<math.h>

void init_array(float *a, const int N);
__global__
void sum(float* input)
{
 const int tid = threadIdx.x;
 int no_threads = blockDim.x;

 int step_size =1;

 while(no_threads>0)
 {
   if(tid<no_threads)
   {
      const int fst = tid*step_size*2;
      const int snd = fst + step_size;
      input[fst] += input[snd];
   }
   step_size <<=1;   //increase by 2
   no_threads >>= 1;  //decrease by 2

 }
}

__global__
void min(float* input)
{
 const int tid = threadIdx.x;
 int no_threads = blockDim.x;

 int step_size =1;

 while(no_threads>0)
 {
   if(tid<no_threads)
   {
      const int fst = tid*step_size*2;
      const int snd = fst + step_size;
      if(input[snd]<input[fst])
         input[fst]  = input[snd];
   }
   step_size <<=1;
   no_threads >>= 1;

 }


}

__global__
void max(float* input)
{
 const int tid = threadIdx.x;
 int no_threads = blockDim.x;

 int step_size =1;

 while(no_threads>0)
 {
   if(tid<no_threads)
   {
      const int fst = tid*step_size*2;
      const int snd = fst + step_size;
      if(input[snd]>input[fst])
         input[fst]  = input[snd];
   }
   step_size <<=1;
   no_threads >>= 1;

 }


}


__global__
void std_(float* input,float avg)
{
 const int tid = threadIdx.x;
 int no_threads = blockDim.x;

 int step_size =1;

 while(no_threads>0)
 {
   if(tid<no_threads)
   {
      const int fst = tid*step_size*2;
      const int snd = fst + step_size;
      input[fst] = (input[fst]-avg)*(input[fst]-avg);
      input[snd] = (input[snd]-avg)*(input[snd]-avg);
      input[fst] += input[snd];
   }
   step_size <<=1;
   no_threads >>= 1;

 }


}


int main()
{

 srand(time(NULL));
// This random generation is referred to as a pseudo-random generation.  These created values are not truly "random" because a mathematical formula is used to generate the values.
//makes use of the computer's internal clock to control the choice of the seed.  Since time is continually changing, the seed is forever changing.  Remember, if the seed number remains the same, the sequence of numbers will be repeated for each run of the program.
 const int N = 8;

 const int size = N*sizeof(float);

 float *a;		//CPU Pointer

 float *d_a,*d_b;	//GPU Pointer

 float result, avg;

 double time_taken;
 
   //A is cpu pointer so memory is allocated by malloc.
 a = (float*)malloc(sizeof(float)*N);

 //initialising the array
 init_array(a,N);

 //printing the array
 for(int i=0;i<N;i++)
   printf("%f   ",a[i]);

//Allocating memory to GPU
 hipMalloc(&d_a,size);
//Copying contents of CPU to GPU
 hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);

 hipMalloc(&d_b,size);
 hipMemcpy(d_b,a,size,hipMemcpyHostToDevice);
 
 //----------------Sum--------------------------
 clock_t t;
//The clock() function returns the approximate processor time that is consumed by the program.
 t = clock();
//Launching Kernel for sum
 sum<<<1,N/2>>>(d_a);
 hipMemcpy(&result,d_a,sizeof(float),hipMemcpyDeviceToHost);
 t = clock() - t;
//formula
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds

 printf("   Sum:  %f",result);
 printf(" Time taken by sum :%f",time_taken);
    
 //----------------Min--------------------------
 t = clock();
 min<<<1,N/2>>>(d_a);
 hipMemcpy(&result,d_a,sizeof(float),hipMemcpyDeviceToHost);
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf("   Min:  %f",result);
 printf(" Time taken by min :%f",time_taken);
      
 //----------------Max--------------------------
 t = clock();
 max<<<1,N/2>>>(d_a); 
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf(" Time taken by max :%f",time_taken);
 hipMemcpy(&result,d_a,sizeof(float),hipMemcpyDeviceToHost);
 printf("   Max:  %f",result);
 printf(" Time taken by max :%f",time_taken);
    
 //----------------Average--------------------
 t = clock();
 sum<<<1,N/2>>>(d_a);
 hipMemcpy(&result,d_a,sizeof(float),hipMemcpyDeviceToHost);
 avg = result/N;
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf("   Avg:  %f",avg);
 printf(" Time taken by avg :%f",time_taken);
 
 
    
 //----------------Standard deviation-------------
  t = clock();
 std_<<<1,N/2>>>(d_a,avg);
 float std;
 hipMemcpy(&std,d_b,sizeof(float),hipMemcpyDeviceToHost);
 std =std/N;
 std = sqrt(std);
 t = clock() - t;
 time_taken = ((double)t)/CLOCKS_PER_SEC; //in seconds
 printf(" STD IS:%f",std);
 printf(" Time taken by std :%f",time_taken);
 

 
 hipFree(d_a);
 hipFree(d_b);
 delete[] a;

 return 0;
}


void init_array(float*a,const int N)
{
  for(int i=0;i<N;i++)
     a[i] = rand()%N + 1;
}
